#include "hip/hip_runtime.h"



/*
__global__ void compute_forces_subset(CUDA_REAL* result, const CUDA_REAL* ptcl, const CUDA_REAL *diff, const CUDA_REAL* magnitudes, int n, int m, const int* subset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//if (idx < m * n) {
	int i = subset[idx / n];
	int j = idx % n;
	int six_idx = idx*_six;
	CUDA_REAL scale;
	idx *= _two;
	__shared__ CUDA_REAL res[_six];

	if (threadIdx.x == 0) { 
		res[0]=0;
		res[1]=0;
		res[2]=0;
		res[3]=0;
		res[4]=0;
		res[5]=0;
	}

	if (idx >= m * n || i == j || magnitudes[idx] <= 0.) {
		atomicAdd(&res[0], 0.);
		atomicAdd(&res[1], 0.);
		atomicAdd(&res[2], 0.);

		atomicAdd(&res[3], 0.);
		atomicAdd(&res[4], 0.);
		atomicAdd(&res[5], 0.);
	}
	else  {
		scale = ptcl[_seven*j+6] / (magnitudes[idx] *sqrtf(magnitudes[idx]));
		i *= _six;
		atomicAdd(&res[0], scale * diff[six_idx]);
		atomicAdd(&res[1], scale * diff[six_idx + 1]);
		atomicAdd(&res[2], scale * diff[six_idx + 2]);

		atomicAdd(&res[3], scale * (diff[six_idx + 3] - magnitudes[idx+1]*diff[six_idx    ]/magnitudes[idx]));
		atomicAdd(&res[4], scale * (diff[six_idx + 4] - magnitudes[idx+1]*diff[six_idx + 1]/magnitudes[idx]));
		atomicAdd(&res[5], scale * (diff[six_idx + 5] - magnitudes[idx+1]*diff[six_idx + 2]/magnitudes[idx]));
	}
	__syncthreads();

	if (threadIdx.x == 0) { 
		result[i]   = res[0];
		result[i+1] = res[1];
		result[i+2] = res[2];
		result[i+3] = res[3];
		result[i+4] = res[4];
		result[i+5] = res[5];
	}
}
*/




struct CudaParticle{
  float3 pos;
  float3 vel;
  float  mass;
	float  r2; // for AC neighbor
	float  mdot;

	//BackgroundParticle(int) {}
	CudaParticle(float m, float x[3], float v[3], float _r2, float _mdot){
		mass  = m;
		pos.x = x[0];
    pos.y = x[1];
    pos.z = x[2];
    vel.x = v[0];
    vel.y = v[1];
    vel.z = v[2];
		mdot  = _mdot;
		r2  = _r2;

    NAN_CHECK(x[0]);
    NAN_CHECK(x[1]);
    NAN_CHECK(x[2]);
    NAN_CHECK(m);
    NAN_CHECK(v[0]);
    NAN_CHECK(v[1]);
    NAN_CHECK(v[2]);
    NAN_CHECK(_mdot);
    NAN_CHECK(_r2);
  }

  void setParticle(float m, float x[3], float v[3], float _r2, float _mdot){
		mass  = m;
		pos.x = x[0];
		pos.y = x[1];
		pos.z = x[2];
		vel.x = v[0];
		vel.y = v[1];
		vel.z = v[2];
		mdot  = _mdot;
		r2  = _r2;

		NAN_CHECK(x[0]);
		NAN_CHECK(x[1]);
		NAN_CHECK(x[2]);
		NAN_CHECK(m);
		NAN_CHECK(v[0]);
		NAN_CHECK(v[1]);
		NAN_CHECK(v[2]);
		NAN_CHECK(_mdot);
    NAN_CHECK(_r2);
	}
  //__device__ BackgroundParticle() {}
};



struct Result{
	float3 acc;
	float3 adot;
	//unsigned short num_ac;          //  8 words
	//unsigned short ac_list[MaxNeighbor];

	void clear_h(void) {
		acc.x  = acc.y  = acc.z  = 0.f;
		adot.x = adot.y = adot.z = 0.f;
		//nnb = 0;
	}

	__device__  void clear() {
		acc.x  = acc.y  = acc.z  = 0.f;
		adot.x = adot.y = adot.z = 0.f;
	}

	/*
	__device__ void operator+=(const Result &rhs){
		acc.x  += rhs.acc.x;
		acc.y  += rhs.acc.y;
		acc.z  += rhs.acc.z;
		adot.x += rhs.adot.x;
		adot.y += rhs.adot.y;
		adot.z += rhs.adot.z;
	}
	*/
};

struct  Neighbor{
	int NumNeighbor;
	int NeighborList[NumNeighborMax]; // this needs to be modified.


	__device__ void clear() {
		NumNeighbor = 0;
#pragma unroll
		for (int i=0; i<NumNeighborMax; i++) {
			NeighborList[i] = 0;
		}
	}
	void clear_h() {
		NumNeighbor = 0;
#pragma unroll
		for (int i=0; i<NumNeighborMax; i++) {
			NeighborList[i] = 0;
		}
	}
};


/*
struct  Neighbor_d{
	int NumNeighbor;
	int NeighborList[100]; // this needs to be modified.


	__device__ void clear() {
		NumNeighbor = 0;
#pragma unroll
		for (int i=0; i<100; i++) {
			NeighborList[i] = 0;
		}
	}
	void clear_h() {
		NumNeighbor = 0;
#pragma unroll
		for (int i=0; i<100; i++) {
			NeighborList[i] = 0;
		}
	}
};
*/

/*
struct  Neighbor{
	int width = 2;
	int height = 100;

	int NumNeighbor[2];
	int NeighborList[2][100]; // this needs to be modified.

	int* NeighborList_d;
	int* NumNeighbor_d;
	size_t pitch;

  Neighbor(){
		hipError_t cudaStatus;
		cudaStatus = hipMallocPitch(&NeighborList_d, &pitch, width * sizeof(int), height);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor hipMallocPitch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipMalloc(&NumNeighbor_d, width * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
		}
  }
	void toHost() {
		hipError_t cudaStatus;
		cudaStatus = hipMemcpy(NumNeighbor, NumNeighbor_d, width * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipMemcpy2D(NeighborList, width * sizeof(int), NeighborList_d, pitch, width * sizeof(int), height, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor cudaMemcpy2d failed: %s\n", hipGetErrorString(cudaStatus));
		}
	}

	void toDevice() {
		hipError_t cudaStatus;
		cudaStatus = hipMemcpy(NumNeighbor_d, NumNeighbor, width * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy2D(NeighborList_d, pitch, NeighborList, width * sizeof(int), width * sizeof(int), height, hipMemcpyHostToDevice);
	}

	void free() {
		hipError_t cudaStatus;
		cudaStatus = hipFree(NeighborList_d);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor hipFree list failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipFree(NumNeighbor_d);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Neighbor hipFree num failed: %s\n", hipGetErrorString(cudaStatus));
		}
	}
};
*



