#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <cmath>
#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../defs.h"
#include "cuda_defs.h"
#include "cuda_kernels.h"
#include "cuda_routines.h"

#ifdef NSIGHT
#include <nvToolsExt.h>
#endif



static int NNB;
static CUDA_REAL time_send, time_grav, time_out, time_nb;
static long long numInter;
static int icall,ini,isend;
static int nbodymax;


static int devid, numGPU;
static bool is_open = false;
static bool devinit = false;
static bool first   = true;
static int variable_size;
static int target_size;

extern CUDA_REAL *h_ptcl, *d_ptcl; //, *background;
extern CUDA_REAL *h_result, *d_result;
extern CUDA_REAL *d_diff, *d_magnitudes, *d_r2;
extern int *d_target;

CUDA_REAL *h_ptcl=nullptr, *d_ptcl=nullptr;; //, *background;
CUDA_REAL *h_result=nullptr, *d_result=nullptr;
CUDA_REAL *d_diff=nullptr,*d_magnitudes=nullptr, *d_r2=nullptr;
int *d_target=nullptr;

#define TEST_CUBLAS
#ifndef TEST_CUBLAS
extern int *h_neighbor, *d_neighbor, *h_num_neighbor, *d_num_neighbor;
int *h_neighbor=nullptr, *d_neighbor=nullptr, *d_num_neighbor=nullptr, *h_num_neighbor=nullptr;

#else
extern bool *h_neighbor, *d_neighbor;
extern int *h_num_neighbor;
bool *h_neighbor=nullptr, *d_neighbor=nullptr;
int *h_num_neighbor=nullptr; // added by wispedia
#endif

extern hipStream_t stream;
hipStream_t stream;

extern CUDA_REAL *h_diff, *h_magnitudes;
CUDA_REAL *h_diff, *h_magnitudes;



/*************************************************************************
 *	 Computing Acceleration
 *************************************************************************/

void GetAcceleration(
		int NumTargetTotal,
		int h_target_list[],
		CUDA_REAL acc[][3],
		CUDA_REAL adot[][3],
		int NumNeighbor[],
		int **NeighborList
		) {

	assert(is_open);
	assert((NumTargetTotal > 0) && (NumTargetTotal <= NNB));

	int minGridSize, blockSize, gridSize;
	int sharedMemSize;

	//hipStreamCreate(&stream);

	hipblasHandle_t handle;
	initializeCudaAndCublas(&handle);
	
	/*
	for(int i=0; i<NumTarget; i++) {
		d_result[i].clear();
		d_neighbor[i].clear();
		d_dist = 0.;
	}
	*/
	/*
	fprintf(stderr,"\ntargets=");
	for(int i=0; i<NumTarget; i++) {
		fprintf(stderr,"%d, ", h_target_list[i]);
	}
	fprintf(stderr,"\n");
	*/
	int total_data_num;
	int NumTarget;
	toDevice(h_target_list, d_target, NumTargetTotal, stream);

	for (int TargetStart=0; TargetStart < NumTargetTotal; TargetStart+=target_size){
		NumTarget = std::min(target_size, NumTargetTotal-TargetStart);
		// fprintf(stdout, "TargetStart=%d, NumTargetTotal=%d, NumTarget=%d\n", TargetStart, NumTargetTotal, NumTarget); // Eunwoo debug


		// Compute pairwise differences for the subset
		//blockSize = variable_size;
		//gridSize = NumTarget;
		total_data_num = new_size(NNB*NumTarget);
		/******* Initialize *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					initialize, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		initialize<<<gridSize, blockSize, 0, stream>>>\
			(d_result, d_diff, d_magnitudes, NNB, NumTarget, d_target);
		hipDeviceSynchronize();


		/******* Differencese *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_pairwise_diff_subset, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_pairwise_diff_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_ptcl, d_diff, NNB, NumTarget, d_target, TargetStart);
		hipDeviceSynchronize();

		/******* Magnitudes *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_magnitudes_subset, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_magnitudes_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_r2, d_diff, d_magnitudes, NNB, NumTarget, d_target, d_neighbor, TargetStart);
		hipDeviceSynchronize();

		/******* Force *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_forces_subset, 0, 0));
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_forces_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_ptcl, d_diff, d_magnitudes, NNB, NumTarget, d_target);



		/******* Neighborhood (new) *********/
		// reduce_neighbors(handle, d_neighbor, d_num_neighbor, d_magnitudes, NNB, NumTarget, h_target_list);
		hipDeviceSynchronize();

		#ifdef NSIGHT
		nvtxRangePushA("Reduction");
		#endif
		/******* Reduction *********/
		reduce_forces_cublas(handle, d_diff, d_result, NNB, NumTarget); //test by wispedia
		//reduce_forces_thrust(d_diff, d_result, NNB, NumTarget);
		hipDeviceSynchronize();

		#ifdef NSIGHT
		nvtxRangePop();
		#endif


		//print_forces_subset<<<gridSize, blockSize>>>\
			(d_result, NumTarget);	


		hipStreamSynchronize(stream); // Wait for all operations to finish
		toHost(h_result + _six * TargetStart, d_result, _six * NumTarget);

		#ifdef NSIGHT
		nvtxRangePushA("Neighbor in CPU");
		#endif


		toHost(h_neighbor, d_neighbor, NNB * NumTarget);
		for (int i=0;i<NumTarget;i++) {
			int k = 0;
			int* targetNeighborList = NeighborList[i + TargetStart]; // Cache the row pointer
			int target = h_target_list[i + TargetStart]; // Cache the target value

			for (int j=0;j<NNB;j++) {
				if (h_neighbor[i * NNB + j] && (target != j)) {
					if (k<NumNeighborMax){
						targetNeighborList[k++] = j;
						}
					else {
						fprintf(stderr, "Number of neighbors exceeds the maximum number of neighbors %d\n", k);
						exit(1);
						}
				}
			}
			NumNeighbor[i + TargetStart] = k; // h_num_neighbor[i];

		}
		#ifdef NSIGHT
		nvtxRangePop();
		#endif

	}

	// out data
	for (int i=0; i<NumTargetTotal; i++) {
		acc[i][0]  = h_result[_six*i];
		acc[i][1]  = h_result[_six*i+1];
		acc[i][2]  = h_result[_six*i+2];
		adot[i][0] = h_result[_six*i+3];
		adot[i][1] = h_result[_six*i+4];
		adot[i][2] = h_result[_six*i+5];

		/*
		fprintf(stderr, "%d (%d) neighbors of %d = ", i, h_target_list[i], NumNeighbor[i]);
		for (int j=0;j<NumNeighbor[i];j++) {
			fprintf(stderr, "%d, ", NeighborList[i][j]);
		}
		fprintf(stderr, "\n");
		*/
	}


	hipblasDestroy(handle);
	/*
	my_free(h_background , d_background);
	my_free(h_result     , d_result);
	my_free(h_target     , d_target);
	my_free(h_neighbor   , d_neighbor);
	*/
	//hipStreamDestroy(stream);
	//my_free_d(do_neighbor);
	//printf("CUDA: done?\n");
}







/*************************************************************************
 *	 Communication with HOST
 *************************************************************************/
void _ReceiveFromHost(
		int _NNB,
		CUDA_REAL m[],
		CUDA_REAL x[][3],
		CUDA_REAL v[][3],
		CUDA_REAL r2[],
		CUDA_REAL mdot[]
		){
	//time_send -= get_wtime();
	nbodymax       = 100000000;
	NNB            = _NNB;
	//NumNeighborMax = _NumNeighborMax;
	isend++;
	assert(NNB <= nbodymax);
	hipError_t cudaStatus;

	//printf("CUDA: receive starts\n");
	//my_allocate(&h_background, &d_background_tmp, new_size(NNB));
	//hipMemcpyToSymbol(HIP_SYMBOL(d_background), &d_background_tmp, new_size(NNB)*sizeof(BackgroundParticle));


	if ((first) || (new_size(NNB) > variable_size )) {
		variable_size = new_size(NNB);
		target_size = ((NNB > nbodymax/NNB) ? int(pow(2,ceil(log(nbodymax/NNB)/log(2.0)))) : NNB);
		fprintf(stderr, "variable_size=%d, target_size=%d\n", variable_size, target_size);

		if (!first) {
			my_free(h_ptcl				 , d_ptcl);
			my_free(h_result       , d_result);
			my_free(h_neighbor     , d_neighbor);
			// my_free(h_num_neighbor , d_num_neighbor);
			hipHostFree(h_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			hipFree(d_magnitudes);

		}
		else {
			first = false;
		}
		my_allocate(&h_ptcl         , &d_ptcl        ,         _seven*variable_size); // x,v,m
		my_allocate(&h_result       , &d_result      ,           _six*variable_size);
		// my_allocate(&h_num_neighbor , &d_num_neighbor,                variable_size);
		// my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		hipMalloc((void**)&d_r2        ,        variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_target    ,        variable_size * sizeof(int));
		hipMalloc((void**)&d_diff      , _six * variable_size * target_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_magnitudes, _two * variable_size * target_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_diff          , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_magnitudes    , _two * variable_size * variable_size * sizeof(CUDA_REAL));
		#ifdef TEST_CUBLAS
		my_allocate(&h_neighbor     , &d_neighbor    , variable_size * target_size);
		hipHostMalloc((void**)&h_num_neighbor, variable_size * sizeof(int));
		#endif
		
	}


	for (int j=0; j<NNB; j++) {
		for (int dim=0; dim<Dim; dim++) {
			h_ptcl[_seven*j+dim]   = x[j][dim];
			h_ptcl[_seven*j+dim+3] = v[j][dim];
		}
		h_ptcl[_seven*j+6] = m[j];
		//h_particle[j].setParticle(m[j], x[j], v[j], r2[j], mdot[j]);
	}

	//toDevice(h_background,d_background,variable_size);
	toDevice(h_ptcl,d_ptcl, _seven*NNB, stream);
	toDevice(r2    ,d_r2  ,        NNB, stream);
	//fprintf(stdout, "CUDA: receive done\n");
}



void _InitializeDevice(int irank){

	std::cout << "Initializing CUDA ..." << std::endl;
	// Select CUDA device (optional)
	int device = 0; // Choose GPU device 0
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	hipStreamCreate(&stream);

	std::cout << "There are " << deviceCount << " GPUs." << std::endl;
	if (device < 0 || device >= deviceCount) {
		    // Handle invalid device index
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");


	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	

	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);


	hipSetDevice(device);

	// Initialize CUDA context
	/*
	hipError_t cudaStatus = hipFree(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
		return;
	}
	*/

	is_open = true;
	// CUDA is now initialized and ready to be used
	std::cout << "CUDA initialized successfully!" << std::endl;

	/*
	if(devinit) return;

	hipGetDeviceCount(&numGPU);
	assert(numGPU > 0);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list)
	{
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		if (p) {
			devid = atoi(p);
			numGPU++;
		}
		assert(numGPU > 0);
	}else{
		devid=irank%numGPU;
	}
	hipSetDevice(devid);

#ifdef PROFILE
	//  if(!irank)fprintf(stderr, "***********************\n");
	//  if(!irank)fprintf(stderr, "Initializing NBODY6/GPU library\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");
	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);
	//  if(!irank)fprintf(stderr, "***********************\n");
#endif
	devinit = true;
	*/
}



void _OpenDevice(const int irank){
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend = 0;

	//select GPU========================================//
	_InitializeDevice(irank);

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


#ifdef PROFILE
	//	fprintf(stderr, "RANK: %d ******************\n",irank);
	//	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "# Open GPU regular force - rank: %d\n", irank);
	//fprintf(stderr, "***********************\n");
#endif
}



void _CloseDevice() {
	if(!is_open) {
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;


	hipError_t error;

	printf("CUDA: ?!! ...\n");
	//my_free(&h_result    , &d_result);
	fprintf(stderr, "result ...\n");
	//my_free(&h_target    , &d_target);
	fprintf(stderr, "target ...\n");
	//my_free(&h_neighbor  , &d_neighbor);
	fprintf(stderr, "neighbor ...\n");
	//my_free(&h_background, &d_background);

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		// Handle error
	}

#ifdef PROFILE
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "rank: %d***************\n",devid);
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "time nb   : %f sec\n", time_nb);
	fprintf(stderr, "time out  : %f sec\n", time_out);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}



void _ProfileDevice(int irank) {
#ifdef PROFILE
	if(icall) {
		fprintf(stderr,"[R.%d-D.%d GPU Reg.F ] Nsend %d  Ngrav %d  <Ni> %d   send(s) %f grav(s) %f  nb(s) %f  out(s) %f  Perf.(Gflops) %f\n",irank,devid,isend,icall,ini/isend,time_send,time_grav,time_nb,time_out,60.e-9*numInter/time_grav);
	}
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend= 0;
#else
	return;
#endif
}


#define mexPrintf printf

inline void gpuMemReport(size_t * avail, size_t * total, 
		        const char * title = 0, const size_t * free = 0, const bool sense = true) 
{
	char tstring[32] = { '\0' };
	hipMemGetInfo(avail, total);  

	if (free) {
		if (title) {
			strncpy(tstring, title, 31);
		}
		mexPrintf("%s Memory avaliable: Free: %zu, Total: %zu, %s: %zu\n",
				tstring, *avail, *total, (sense) ? "Allocated\0" : "Freed\0", 
				(sense) ? (*free - *avail) : (*avail - *free));
	} else {
		mexPrintf("Memory avaliable: Free: %zu, Total: %zu\n", *avail, *total);  
	}
}



extern "C" {
	void InitializeDevice(int *irank){
		_InitializeDevice(*irank);
	}
	void OpenDevice(const int *irank){
		_OpenDevice(*irank);
	}
	void CloseDevice(){
		_CloseDevice();
	}
	void SendToDevice(int *_NNB, CUDA_REAL m[], CUDA_REAL x[][3], CUDA_REAL v[][3], CUDA_REAL r2[], CUDA_REAL mdot[]) {
		_ReceiveFromHost(*_NNB, m, x, v, r2, mdot);
	}
	void ProfileDevice(int *irank){
		_ProfileDevice(*irank);
	}
	void CalculateAccelerationOnDevice(int *NumTargetTotal, int *h_target_list, CUDA_REAL acc[][3], CUDA_REAL adot[][3], int NumNeighbor[], int **NeighborList) {
		GetAcceleration(*NumTargetTotal, h_target_list, acc, adot, NumNeighbor, NeighborList);
	}
}

