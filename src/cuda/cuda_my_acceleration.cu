#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <cmath>
#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../defs.h"
#include "cuda_defs.h"
#include "cuda_kernels.h"
#include "cuda_routines.h"

#ifdef NSIGHT
#include <nvToolsExt.h>
#endif



static int NNB;
static CUDA_REAL time_send, time_grav, time_out, time_nb;
static long long numInter;
static int icall,ini,isend;
static int nbodymax;


static int devid, numGPU;
static bool is_open = false;
static bool devinit = false;
static bool first   = true;
static int variable_size;
static int target_size;

extern CUDA_REAL *h_ptcl, *d_ptcl; //, *background;
extern CUDA_REAL *h_result, *d_result;
extern CUDA_REAL *d_diff, *d_magnitudes, *d_r2;
extern int *d_target;

CUDA_REAL *h_ptcl=nullptr, *d_ptcl=nullptr;; //, *background;
CUDA_REAL *h_result=nullptr, *d_result=nullptr;
CUDA_REAL *d_diff=nullptr,*d_magnitudes=nullptr, *d_r2=nullptr;
int *d_target=nullptr;

<<<<<<< HEAD
#define TEST_CUBLAS
#ifndef TEST_CUBLAS
extern int *h_neighbor, *d_neighbor, *h_num_neighbor, *d_num_neighbor;
int *h_neighbor=nullptr, *d_neighbor=nullptr, *d_num_neighbor=nullptr, *h_num_neighbor=nullptr;

#else
extern bool *h_neighbor, *d_neighbor;
extern int *h_num_neighbor;
bool *h_neighbor=nullptr, *d_neighbor=nullptr;
int *h_num_neighbor=nullptr; // added by wispedia
#endif
=======
#ifdef debuggig_verification
extern CUDA_REAL *h_r2;
CUDA_REAL *h_r2=nullptr; //only for verification
#endif

extern int *h_neighbor, *d_neighbor;
extern int *h_num_neighbor, *d_num_neighbor, *d_neighbor_block;
int *h_neighbor=nullptr, *d_neighbor=nullptr;
int *h_num_neighbor=nullptr, *d_num_neighbor=nullptr; // added by wispedia
int *d_neighbor_block=nullptr; // added by wispedia
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3

extern hipStream_t stream;
hipStream_t stream;

extern CUDA_REAL *h_diff, *h_magnitudes;
CUDA_REAL *h_diff, *h_magnitudes;


/*************************************************************************
 *	 Computing Acceleration
 *************************************************************************/

void GetAcceleration(
		int NumTargetTotal,
		int h_target_list[],
		CUDA_REAL acc[][3],
		CUDA_REAL adot[][3],
		int NumNeighbor[],
		int *NeighborList
		) {

	assert(is_open);
	assert((NumTargetTotal > 0) && (NumTargetTotal <= NNB));

	int minGridSize, blockSize, gridSize;
	int sharedMemSize;

	//hipStreamCreate(&stream);

	hipblasHandle_t handle;
	initializeCudaAndCublas(&handle);
	
	/*
	for(int i=0; i<NumTarget; i++) {
		d_result[i].clear();
		d_neighbor[i].clear();
		d_dist = 0.;
	}
	*/
	/*
	fprintf(stderr,"\ntargets=");
	for(int i=0; i<NumTarget; i++) {
		fprintf(stderr,"%d, ", h_target_list[i]);
	}
	fprintf(stderr,"\n");
	*/
	int total_data_num;
	int NumTarget;
	toDevice(h_target_list, d_target, NumTargetTotal, stream);

	for (int TargetStart=0; TargetStart < NumTargetTotal; TargetStart+=target_size){
		NumTarget = std::min(target_size, NumTargetTotal-TargetStart);
		// fprintf(stdout, "TargetStart=%d, NumTargetTotal=%d, NumTarget=%d\n", TargetStart, NumTargetTotal, NumTarget); // Eunwoo debug


		// Compute pairwise differences for the subset
		//blockSize = variable_size;
		//gridSize = NumTarget;
		total_data_num = new_size(NNB*NumTarget);
		/******* Initialize *********/
<<<<<<< HEAD
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					initialize, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		initialize<<<gridSize, blockSize, 0, stream>>>\
			(d_result, d_diff, d_magnitudes, NNB, NumTarget, d_target);
		hipDeviceSynchronize();


		/******* Differencese *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_pairwise_diff_subset, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_pairwise_diff_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_ptcl, d_diff, NNB, NumTarget, d_target, TargetStart);
		hipDeviceSynchronize();

		/******* Magnitudes *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_magnitudes_subset, 0, 0));	
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_magnitudes_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_r2, d_diff, d_magnitudes, NNB, NumTarget, d_target, d_neighbor, TargetStart);
		hipDeviceSynchronize();

		/******* Force *********/
		checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
					compute_forces_subset, 0, 0));
		gridSize = (total_data_num + blockSize - 1) / blockSize;

		compute_forces_subset<<<gridSize, blockSize, 0, stream>>>\
			(d_ptcl, d_diff, d_magnitudes, NNB, NumTarget, d_target);



		/******* Neighborhood (new) *********/
		// reduce_neighbors(handle, d_neighbor, d_num_neighbor, d_magnitudes, NNB, NumTarget, h_target_list);
		hipDeviceSynchronize();

=======

		// checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initialize, 0, 0));	
		// gridSize = (new_size(NumTarget*GridDimY) + blockSize - 1) / blockSize;
		// initialize<<<gridSize, blockSize, 0, stream>>>(d_result, d_diff, GridDimY, NumTarget, d_target);
		// hipDeviceSynchronize();
		


		dim3 blockDim(64, 1, 1);  // Use a 1D block with 256 threads
		dim3 gridDim((NumTarget + BatchSize + blockDim.x - 1) / blockDim.x, GridDimY);
		// dim3 gridDim(32, 32, 1);    // Adjust grid size as needed
		printf("blockDim=(%d, %d), gridDim=(%d, %d)\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);
		compute_forces<<<gridDim, blockDim, 0, stream>>>\
			(d_ptcl, d_r2, d_diff, NumTarget, NNB, d_target, d_neighbor_block, d_num_neighbor, TargetStart);

		hipDeviceSynchronize();

		/******* Reduction *********/
		reduce_forces_cublas(handle, d_diff, d_result, GridDimY, NumTarget);
		//reduce_forces_thrust(d_diff, d_result, NNB, NumTarget);
		hipDeviceSynchronize();


		#define newGather
		#ifdef newGather

		#ifdef NSIGHT
		nvtxRangePushA("Reduction");
		#endif

		// blockSize = 64;
		// gridSize = (total_data_num + blockSize - 1) / blockSize;
		// print_forces_subset<<<gridSize, blockSize>>>(d_result, NumTarget);

		dim3 gridDim2(NumTarget, 1);
		dim3 blockDim2(GridDimY, 1);  // Use a 1D block with 256 threads

		printf("blockDim=(%d, %d), gridDim=(%d, %d)\n", blockDim2.x, blockDim2.y, gridDim2.x, gridDim2.y);
		gather_neighbor<<<gridDim2, blockDim2, 0, stream>>>\
			(d_neighbor_block, d_num_neighbor, d_neighbor, NumTarget);

		#ifdef NSIGHT
		nvtxRangePop();
		#endif

		hipStreamSynchronize(stream); // Wait for all operations to finish
		toHost(h_result + _six * TargetStart, d_result, _six * NumTarget);

		#ifdef NSIGHT
		nvtxRangePushA("Neighbor in CPU1");
		#endif

		// toHost(NeighborList, d_neighbor, NumTarget * NumNeighborMax);
		toHost(NeighborList, d_neighbor, NumTarget * NumNeighborMax);


		#ifdef NSIGHT
		nvtxRangePop();
		#endif
		// toHost(h_neighbor, d_neighbor, NumTarget * NumNeighborMax);//
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3
		#ifdef NSIGHT
		nvtxRangePushA("Neighbor in CPU2");
		#endif

		toHost(h_num_neighbor, d_num_neighbor, NumTarget * GridDimY);
		#ifdef NSIGHT
		nvtxRangePop();
		#endif

		#ifdef NSIGHT
		nvtxRangePushA("Neighbor in CPU3");
		#endif

<<<<<<< HEAD
		//print_forces_subset<<<gridSize, blockSize>>>\
			(d_result, NumTarget);	
=======
		for (int i=0;i<NumTarget;i++) {
	        int k = 0;
			// Loop over each block in the Y dimension
			for (int j = 0; j < GridDimY; j++) {
				k += h_num_neighbor[i * GridDimY + j]; // Number of neighbors in the current block
			}
			NumNeighbor[i + TargetStart] = k;
		}
		#ifdef NSIGHT
		nvtxRangePop();
		#endif
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3

		#else

		hipStreamSynchronize(stream); // Wait for all operations to finish
		toHost(h_result + _six * TargetStart, d_result, _six * NumTarget);

		#ifdef NSIGHT
		nvtxRangePushA("Neighbor in CPU");
		#endif

		toHost(h_neighbor, d_neighbor_block, NumTarget * GridDimY * NNB_per_block);//
		toHost(h_num_neighbor, d_num_neighbor, NumTarget * GridDimY);

		for (int i=0;i<NumTarget;i++) {
			int* targetNeighborList = NeighborList[i + TargetStart]; // Cache the row pointer
<<<<<<< HEAD
			int target = h_target_list[i + TargetStart]; // Cache the target value
=======
	        int k = 0; // Counter for the number of neighbors added
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3

			// Loop over each block in the Y dimension
			for (int j = 0; j < GridDimY; j++) {
				int* blockNeighborList = &h_neighbor[(i * GridDimY + j) * NNB_per_block]; // Pointer to the neighbor list of the current block
				int numNeighborsInBlock = h_num_neighbor[i * GridDimY + j]; // Number of neighbors in the current block

				// Loop over each neighbor in the current block
				for (int n = 0; n < numNeighborsInBlock; n++) {
					if (k < NumNeighborMax){
						targetNeighborList[k++] = blockNeighborList[n];
					}
					else {
						fprintf(stderr, "Number of neighbors exceeds the maximum number of neighbors %d\n", k);
						exit(1);
					}
				}
			}
			// Store the number of neighbors for this target
			NumNeighbor[i + TargetStart] = k;

		}
		#ifdef NSIGHT
		nvtxRangePop();
		#endif
<<<<<<< HEAD
=======
		
		#endif

>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3

	}

	#ifdef NSIGHT
	nvtxRangePushA("Out data");
	#endif
	// out data
	for (int i=0; i<NumTargetTotal; i++) {
		acc[i][0]  = h_result[_six*i];
		acc[i][1]  = h_result[_six*i+1];
		acc[i][2]  = h_result[_six*i+2];
		adot[i][0] = h_result[_six*i+3];
		adot[i][1] = h_result[_six*i+4];
		adot[i][2] = h_result[_six*i+5];
		
		// #define debuggig_verification
		#ifdef debuggig_verification
		toHost(h_r2, d_r2, NNB); // only for verification

		fprintf(stderr, "%d (%d) neighbors of %d = ", i, h_target_list[i], NumNeighbor[i]);
		for (int j=0;j<NumNeighbor[i];j++) {
			fprintf(stderr, "%d, ", NeighborList[i * NumNeighborMax + j]);
		}
		fprintf(stderr, "\n");

		// verification
		fprintf(stderr, "%d (%d) neighbors of %d (veri)= ", i, h_target_list[i], NumNeighbor[i]);
		double ix = h_ptcl[h_target_list[i]];
		double iy = h_ptcl[h_target_list[i] + NNB * 1];
		double iz = h_ptcl[h_target_list[i] + NNB * 2];
		double i_r2 = h_r2[h_target_list[i]];
		
		fprintf(stderr, "h_r2 = %e \n", i_r2);

		for (int j=0; j<NNB; j++) {
			double dx = ix - h_ptcl[j];
			double dy = iy - h_ptcl[j + NNB * 1];
			double dz = iz - h_ptcl[j + NNB * 2];
			double r2_temp = dx*dx + dy*dy + dz*dz;
			if (r2_temp < i_r2) {
				fprintf(stderr, "%d, (%e)", j, r2_temp);
			}
		}
		fprintf(stderr, "\n");
		exit(1);
		#endif
	}

	#ifdef NSIGHT
	nvtxRangePop();
	#endif
	
	hipblasDestroy(handle);
	/*
	my_free(h_background , d_background);
	my_free(h_result     , d_result);
	my_free(h_target     , d_target);
	my_free(h_neighbor   , d_neighbor);
	*/
	//hipStreamDestroy(stream);
	//my_free_d(do_neighbor);
	//printf("CUDA: done?\n");
}







/*************************************************************************
 *	 Communication with HOST
 *************************************************************************/
void _ReceiveFromHost(
		int _NNB,
		CUDA_REAL m[],
		CUDA_REAL x[][3],
		CUDA_REAL v[][3],
		CUDA_REAL r2[],
		CUDA_REAL mdot[]
		){
	//time_send -= get_wtime();
	nbodymax       = 100000000;
	NNB            = _NNB;
	//NumNeighborMax = _NumNeighborMax;
	isend++;
	assert(NNB <= nbodymax);
	hipError_t cudaStatus;

	//printf("CUDA: receive starts\n");
	//my_allocate(&h_background, &d_background_tmp, new_size(NNB));
	//hipMemcpyToSymbol(HIP_SYMBOL(d_background), &d_background_tmp, new_size(NNB)*sizeof(BackgroundParticle));


	if ((first) || (new_size(NNB) > variable_size )) {
		variable_size = new_size(NNB);
		target_size = ((NNB > nbodymax/NNB) ? int(pow(2,ceil(log(nbodymax/NNB)/log(2.0)))) : NNB);
		fprintf(stderr, "variable_size=%d, target_size=%d\n", variable_size, target_size);

		if (!first) {
			my_free(h_ptcl				 , d_ptcl);
			my_free(h_result       , d_result);
<<<<<<< HEAD
			my_free(h_neighbor     , d_neighbor);
			// my_free(h_num_neighbor , d_num_neighbor);
			hipHostFree(h_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			hipFree(d_magnitudes);

=======
			my_free(h_num_neighbor , d_num_neighbor);
			// hipHostFree(h_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			#ifdef newGather
			hipFree(d_neighbor_block);
			my_free(h_neighbor     , d_neighbor);
			#else
			my_free(h_neighbor     , d_neighbor_block);
			#endif
			// hipFree(d_magnitudes);
			// hipFree(d_acc);
			// hipFree(d_adot);
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3
		}
		else {
			first = false;
		}
		my_allocate(&h_ptcl         , &d_ptcl        ,         _seven*variable_size); // x,v,m
		my_allocate(&h_result       , &d_result      ,           _six*variable_size);
		// my_allocate(&h_num_neighbor , &d_num_neighbor,                variable_size);
		// my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		hipMalloc((void**)&d_r2        ,        variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_target    ,        variable_size * sizeof(int));
<<<<<<< HEAD
		hipMalloc((void**)&d_diff      , _six * variable_size * target_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_magnitudes, _two * variable_size * target_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_diff          , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_magnitudes    , _two * variable_size * variable_size * sizeof(CUDA_REAL));
		#ifdef TEST_CUBLAS
		my_allocate(&h_neighbor     , &d_neighbor    , variable_size * target_size);
		hipHostMalloc((void**)&h_num_neighbor, variable_size * sizeof(int));
		#endif
=======
		// hipMalloc((void**)&d_diff      , _six * variable_size * target_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_diff      , _six * GridDimY * target_size * sizeof(CUDA_REAL));

		// my_allocate(&h_neighbor     , &d_neighbor    , variable_size * target_size);
		// my_allocate(&h_neighbor     , &d_neighbor    , Grid * target_size);
		#ifdef newGather
		hipMalloc((void**)&d_neighbor_block, GridDimY * NNB_per_block * target_size * sizeof(int));
		my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax * target_size);
		#else
		my_allocate(&h_neighbor     , &d_neighbor_block    , GridDimY * NNB_per_block * target_size);
		#endif
		my_allocate(&h_num_neighbor , &d_num_neighbor,                GridDimY * variable_size);

		#ifdef debuggig_verification
		hipHostMalloc((void**)&h_r2        ,        variable_size * sizeof(CUDA_REAL)); // only for verification
		#endif debuggig_verification

		// hipHostMalloc((void**)&h_num_neighbor, GridDimY * variable_size * sizeof(int));
		// hipMalloc((void**)&d_num_neighbor, GridDimY * variable_size * sizeof(int));
>>>>>>> f688615c336c02f68febfc5c6d41376999e08ae3
		
	}


	for (int j=0; j<NNB; j++) {
		for (int dim=0; dim<Dim; dim++) {
			h_ptcl[_seven*j+dim]   = x[j][dim];
			h_ptcl[_seven*j+dim+3] = v[j][dim];
		}
		h_ptcl[_seven*j+6] = m[j];
		//h_particle[j].setParticle(m[j], x[j], v[j], r2[j], mdot[j]);
	}

	//toDevice(h_background,d_background,variable_size);
	toDevice(h_ptcl,d_ptcl, _seven*NNB, stream);
	toDevice(r2    ,d_r2  ,        NNB, stream);
	//fprintf(stdout, "CUDA: receive done\n");
}



void _InitializeDevice(int irank){

	std::cout << "Initializing CUDA ..." << std::endl;
	// Select CUDA device (optional)
	int device = 0; // Choose GPU device 0
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	hipStreamCreate(&stream);

	std::cout << "There are " << deviceCount << " GPUs." << std::endl;
	if (device < 0 || device >= deviceCount) {
		    // Handle invalid device index
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");


	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	

	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);


	hipSetDevice(device);

	// Initialize CUDA context
	/*
	hipError_t cudaStatus = hipFree(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA initialization failed: " << hipGetErrorString(cudaStatus) << std::endl;
		return;
	}
	*/

	is_open = true;
	// CUDA is now initialized and ready to be used
	std::cout << "CUDA initialized successfully!" << std::endl;

	/*
	if(devinit) return;

	hipGetDeviceCount(&numGPU);
	assert(numGPU > 0);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list)
	{
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		if (p) {
			devid = atoi(p);
			numGPU++;
		}
		assert(numGPU > 0);
	}else{
		devid=irank%numGPU;
	}
	hipSetDevice(devid);

#ifdef PROFILE
	//  if(!irank)fprintf(stderr, "***********************\n");
	//  if(!irank)fprintf(stderr, "Initializing NBODY6/GPU library\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");
	char hostname[150];
	memset(hostname,0,150);
	gethostname(hostname,150);
	fprintf(stderr, "# GPU initialization - rank: %d; HOST %s; NGPU %d; device: %d %s\n", irank, hostname,numGPU, devid, prop.name);
	//  if(!irank)fprintf(stderr, "***********************\n");
#endif
	devinit = true;
	*/
}



void _OpenDevice(const int irank){
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend = 0;

	//select GPU========================================//
	_InitializeDevice(irank);

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


#ifdef PROFILE
	//	fprintf(stderr, "RANK: %d ******************\n",irank);
	//	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "# Open GPU regular force - rank: %d\n", irank);
	//fprintf(stderr, "***********************\n");
#endif
}



void _CloseDevice() {
	if(!is_open) {
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;


	hipError_t error;

	printf("CUDA: ?!! ...\n");
	//my_free(&h_result    , &d_result);
	fprintf(stderr, "result ...\n");
	//my_free(&h_target    , &d_target);
	fprintf(stderr, "target ...\n");
	//my_free(&h_neighbor  , &d_neighbor);
	fprintf(stderr, "neighbor ...\n");
	//my_free(&h_background, &d_background);

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		// Handle error
	}

#ifdef PROFILE
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "rank: %d***************\n",devid);
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "time nb   : %f sec\n", time_nb);
	fprintf(stderr, "time out  : %f sec\n", time_out);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}



void _ProfileDevice(int irank) {
#ifdef PROFILE
	if(icall) {
		fprintf(stderr,"[R.%d-D.%d GPU Reg.F ] Nsend %d  Ngrav %d  <Ni> %d   send(s) %f grav(s) %f  nb(s) %f  out(s) %f  Perf.(Gflops) %f\n",irank,devid,isend,icall,ini/isend,time_send,time_grav,time_nb,time_out,60.e-9*numInter/time_grav);
	}
	time_send = time_grav = time_nb = time_out = 0.0;
	numInter = 0;
	icall = ini = isend= 0;
#else
	return;
#endif
}


#define mexPrintf printf

inline void gpuMemReport(size_t * avail, size_t * total, 
		        const char * title = 0, const size_t * free = 0, const bool sense = true) 
{
	char tstring[32] = { '\0' };
	hipMemGetInfo(avail, total);  

	if (free) {
		if (title) {
			strncpy(tstring, title, 31);
		}
		mexPrintf("%s Memory avaliable: Free: %zu, Total: %zu, %s: %zu\n",
				tstring, *avail, *total, (sense) ? "Allocated\0" : "Freed\0", 
				(sense) ? (*free - *avail) : (*avail - *free));
	} else {
		mexPrintf("Memory avaliable: Free: %zu, Total: %zu\n", *avail, *total);  
	}
}



extern "C" {
	void InitializeDevice(int *irank){
		_InitializeDevice(*irank);
	}
	void OpenDevice(const int *irank){
		_OpenDevice(*irank);
	}
	void CloseDevice(){
		_CloseDevice();
	}
	void SendToDevice(int *_NNB, CUDA_REAL m[], CUDA_REAL x[][3], CUDA_REAL v[][3], CUDA_REAL r2[], CUDA_REAL mdot[]) {
		_ReceiveFromHost(*_NNB, m, x, v, r2, mdot);
	}
	void ProfileDevice(int *irank){
		_ProfileDevice(*irank);
	}
	void CalculateAccelerationOnDevice(int *NumTargetTotal, int *h_target_list, CUDA_REAL acc[][3], CUDA_REAL adot[][3], int NumNeighbor[], int *NeighborList) {
		GetAcceleration(*NumTargetTotal, h_target_list, acc, adot, NumNeighbor, NeighborList);
	}
}

