#include "../defs.h"
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda_routines.h"





#ifdef unuse
void initializeCudaAndCublas(hipblasHandle_t* handle) {
	hipError_t cudaStat = hipSetDevice(0);
	if (cudaStat != hipSuccess) {
		std::cerr << "hipSetDevice failed!" << std::endl;
		exit(1);
	}

	hipblasStatus_t stat = hipblasCreate(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		std::cerr << "CUBLAS initialization failed!" << std::endl;
		exit(1);
	}
}

void checkCudaError(hipError_t result) {
	if (result != hipSuccess) {
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
		exit(EXIT_FAILURE);
	}
}

template <typename T>
void my_allocate(T **host, T **device, const int size) {
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(device, size*sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
	}
	//*host = (T*)calloc(size, sizeof(T));
	//*host = (T*)malloc(size*sizeof(T));
	/*
		 if (host == NULL) {
		 fprintf(stderr, "Memory allocation failed\n");
		 }
	 */
	//host = new T[size]();
	cudaStatus = hipHostMalloc(host, size*sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipHostMalloc failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

template <typename T>
void my_free(T &host, T &device) {
	hipError_t cudaStatus;
	cudaStatus = hipFree(device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipHostFree(host);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
	}
	//free(host);
}



template <typename T>
void my_allocate_d(T **device, const int size) {
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(device, size*sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

template <typename T>
void my_free_d(T &device) {
	hipError_t cudaStatus;
	cudaStatus = hipFree(device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
	}
}




template <typename T>
void toDevice(T *host, T *device, const int size, hipStream_t &stream) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpyAsync(device, host, size * sizeof(T), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyHostToDevice failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

template <typename T>
void toHost(T *host, T *device, const int size, hipStream_t &stream) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpyAsync(host, device, size * sizeof(T), hipMemcpyDeviceToHost, stream);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed: %s\n", hipGetErrorString(cudaStatus));
	}
}


template <typename T>
void toDevice(T *host, T *device, const int size) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(device, host, size * sizeof(T), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyHostToDevice failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

template <typename T>
void toHost(T *host, T *device, const int size) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(host, device, size * sizeof(T), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed: %s\n", hipGetErrorString(cudaStatus));
	}
}
#endif


